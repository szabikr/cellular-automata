#include "hip/hip_runtime.h"
#include "CellularAutomata1D.h"
//#include "Rule.h"
#include "Rule.cpp"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

using namespace std;

__constant__ Rule *d_const_rule;

/*
The CUDA kernel for the iteration
*/

__global__ void iteration_kernel(int *state, int state_size, Rule *rule, unsigned int t) {
	rule->setNewState(state, state_size, 0);
}


/*
This method copys the cellular automata and the rule
to the GPU's memory and after that, calls the kernel
to actually make the iteratons on the GPU
*/

void CellularAutomata1D::iterate_gpu(unsigned int t) {

	Rule* d_rule;
	hipMalloc((void**)&d_rule, sizeof(Rule));
	hipMemcpy(d_rule, m_h_rule, sizeof(Rule), hipMemcpyHostToDevice);

	hostRuleTableToDevice(*m_h_rule, *d_rule);

	hipMalloc((void**)&m_d_caState, m_capacity * sizeof(int));
	hipMemcpy(m_d_caState, m_h_caState, m_capacity * sizeof(int), hipMemcpyHostToDevice);
	

	//hipMemcpyToSymbol(HIP_SYMBOL(d_const_rule), m_h_rule, sizeof(Rule));

	cout << "Kernel call.." << endl;
	iteration_kernel<<< 1, 1 >>>(m_d_caState, m_size, d_rule, t);

	hipMemcpy(m_h_caState, m_d_caState, m_capacity * sizeof(int), hipMemcpyDeviceToHost);

}