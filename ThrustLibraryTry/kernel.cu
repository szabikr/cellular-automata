#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>

#include <thrust\device_vector.h>

template <typename T>
class DeviceVector
{
private:
	T* m_bValues;
	std::size_t m_bSize;

public:
	__host__
	void* operator new(std::size_t size)
	{
		DeviceVector<T>* object = nullptr;
		hipMalloc((void**)&object, size);
		return object;
	}

	__host__
	void operator delete(void* object)
	{
		hipFree(object);
	}

	__host__
	DeviceVector(std::size_t size = 1)
	{
		hipMemcpy(&m_bSize, &size, sizeof(std::size_t), hipMemcpyHostToDevice);

		//hipError_t cudaStatus = hipMalloc((void**)m_bValues, sizeof(T*));

		// At this hipMalloc I get Access violation writing location...
		hipMalloc((void**)&m_bValues, size * sizeof(T));

		// It's an alternative solution here
		T* ptr;
		hipMalloc((void**)&ptr, size * sizeof(T));
		hipMemcpy(&m_bValues, &ptr, sizeof(T*), hipMemcpyHostToDevice);
		// The memory is allocated
		// But I can't access it through m_bValues pointer
		// It is also Access violation writing location...
	}

	__host__
	~DeviceVector()
	{
		hipFree(m_bValues);
	}
};

int main()
{
	//DeviceVector<int>* vec = new DeviceVector<int>();

	//delete vec;

	thrust::device_vector<int> vec;

	//std::cout << vec.capacity();

	vec.push_back(1);

	vec.push_back(2);

	vec.push_back(3);

	std::cout << vec.capacity() << std::endl;

	std::cout << vec[3] << std::endl;


	thrust::device_vector<int> vec2(3);


	std::cout << vec2[1] << std::endl;

	return 0;
}


/*

__global__ void myKernelForSize(thrust::device_vector<int>* v, std::size_t* size);
__global__ void myKernelForVecElement(thrust::device_vector<int>* v, int* value);

void sizeFunctionCallFromDeviceCheck();

int main(void)
{
	sizeFunctionCallFromDeviceCheck();
	
	return 0;
}

__global__ void myKernelForSize(thrust::device_vector<int>* v, std::size_t* size)
{
	*size = v->size();
}

__global__ void myKernelForVecElement(thrust::device_vector<int>* v, int* value)
{
	
}

void sizeFunctionCallFromDeviceCheck()
{
	thrust::device_vector<int>* d_vec = new thrust::device_vector<int>(10);
	std::size_t* d_size;
	hipError_t cudaStatus = hipMalloc((void**)&d_size, sizeof(d_size));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Failed at hipMalloc" << std::endl;
	}

	myKernelForSize << <1, 1 >> >(d_vec, d_size);

	std::size_t h_size;

	cudaStatus = hipMemcpy(&h_size, d_size, sizeof(std::size_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Failed at hipMemcpy" << std::endl;
	}

	cudaStatus = hipFree(d_size);
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Failed at hipMalloc" << std::endl;
	}

	delete(d_vec);

	std::cout << "Size: " << h_size << std::endl;
}*/